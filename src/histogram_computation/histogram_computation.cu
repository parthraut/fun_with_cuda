#include "hip/hip_runtime.h"
/*

Problem 4: Histogram Computation
Objective: Compute a histogram of values in an array.

Details:

Given an array A of integers (ranging from 0 to M-1) and its size N, compute the histogram of A. The histogram array H of size M should contain the counts of each integer in A (i.e., H[i] is the number of times i appears in A).
Write a CUDA kernel to compute the histogram in parallel. Consider atomic operations to avoid race conditions when updating the histogram counts.
Discuss and handle the potential performance implications of using atomic operations in global memory.

*/

#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

__global__ void make_histogram(int* A, int* H, int N, int M){
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < N){

        // must be done atomically to avoid race conditions
        atomicAdd(&H[A[index]], 1);
    }
}

int main(int argc, char** argv){

    // array A of integers - size: N, vals: 0 to M-1
    // set N to argv[1]
    int N = atoi(argv[1]);

    int M = 10000;

    int* A = (int*) malloc(N * sizeof(int));

    // generate random values for A
    for(int i = 0; i < N; i++){
        A[i] = rand() % M;
    }

    // create array H of size M
    int* H = (int*) malloc(M * sizeof(int));

    // create device arrays
    int* d_A;
    int* d_H;

    hipMalloc((void**)&d_A, N * sizeof(int));
    hipMalloc((void**)&d_H, M * sizeof(int));

    hipMemset(d_H, 0, M * sizeof(int));

    // copy A to device
    hipMemcpy(d_A, A, N * sizeof(int), hipMemcpyHostToDevice);

    // kernel call
    clock_t start = clock();
    make_histogram<<<(N + 255) / 256, 256>>>(d_A, d_H, N, M);
    hipDeviceSynchronize();
    clock_t end = clock();
    double gpu_time = (double)(end - start) / CLOCKS_PER_SEC;

    // copy H from device
    hipMemcpy(H, d_H, M * sizeof(int), hipMemcpyDeviceToHost);

    // run on cpu to verify
    int* H_cpu = (int*) malloc(M * sizeof(int));
    for(int i = 0; i < M; i++){
        H_cpu[i] = 0;
    }

    start = clock();
    for(int i = 0; i < N; i++){
        H_cpu[A[i]]++;
    }
    end = clock();
    double cpu_time = (double)(end - start) / CLOCKS_PER_SEC;

    // compare results
    bool isValid = true;
    for(int i = 0; i < M; i++){
        if(H[i] != H_cpu[i]){
            isValid = false;
            printf("Error: Histogram mismatch found.\n");
            break;
        }
    }
    if(isValid){
        printf("Success: GPU and CPU histograms match.\n");
    }

    // calculate speedup
    printf("Speedup: %f\n", cpu_time / gpu_time);

    // add gpu_time and cpu_time to a file
    FILE* file = fopen("histogram.csv", "a");
    fprintf(file, "%d, %f, %f, %f\n", atoi(argv[1]), gpu_time, cpu_time, cpu_time / gpu_time);

    // free memory
    free(A);
    free(H);
    free(H_cpu);
    hipFree(d_A);
    hipFree(d_H);

}