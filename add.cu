#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <math.h>

// CUDA Kernel for adding two numbers


__global__ void add_array(int* a, int* b, int* c, int N){
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N) c[index] = a[index] + b[index];

}

void time_gpu_vs_cpu(int n);


int main(){
	time_gpu_vs_cpu(1000);

}



void time_gpu_vs_cpu(int size){

	// host memory pointers
	int* array_a = (int*)malloc(sizeof(int) * size);
	int* array_b = (int*)malloc(sizeof(int) * size);
	int* array_c = (int*)malloc(sizeof(int) * size);

	for(int i = 0; i < size; ++i){
		array_a[i] = array_b[i] = i;
		
	}

	// Time on CPU
	clock_t start = clock();
	for(int i = 0; i < size; ++i){
		array_c[i] = array_a[i] + array_b[i];
	}
	clock_t end = clock();
	double time = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Time on CPU: %f\n", time);
	

	// device memory pointers
	int* array_a_d;
	int* array_b_d;
	int* array_c_d;
	
	hipMalloc((void**)&array_a_d, sizeof(int) * size);
	hipMalloc((void**)&array_b_d, sizeof(int) * size);
	hipMalloc((void**)&array_c_d, sizeof(int) * size);

	// copy
	const int n = size;
	hipMemcpy(array_a_d, array_a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(array_b_d, array_b, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(array_c_d, array_c, n * sizeof(int), hipMemcpyHostToDevice);

	
	int threadsPerBlock = 256;
	int numBlocks = (int)ceil((double)size / (double)threadsPerBlock);

	// Time on GPU
	start = clock();
	add_array<<<numBlocks, threadsPerBlock>>>(array_a_d, array_b_d, array_c_d, size);
	hipDeviceSynchronize();
	end = clock();
	time = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Time on GPU: %f\n", time);

	hipMemcpy(array_c, array_c_d, size, hipMemcpyDeviceToHost);

	printf("success!\n");

	hipFree(array_a_d);
	hipFree(array_b_d);
	hipFree(array_c_d);


	free(array_a);
	free(array_b);
	free(array_c);

}
